#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>

__global__ void de_crossover_kernel(int NP, float CR, float F, int best_model, float* d_ptr, float* d_out_ptr, int size, float* d_all_agent_ids, float* d_Rs, float* d_ris, int layer_idx, int num_layers) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < NP * size) {
		int id = idx / size; // candidate id
		int agent_ids[3]{d_all_agent_ids[id * 3 + 0] * NP, d_all_agent_ids[id * 3 + 1] * NP, d_all_agent_ids[id * 3 + 2] * NP};
		//printf("id: %d, best model: %d, agent 0: %d, agent 1: %d\n", id, best_model, agent_ids[0], agent_ids[1]);
		int R = d_Rs[id] * (float)num_layers;
		float ri = d_ris[layer_idx * NP + id];

		if (ri < CR || layer_idx == R) {
			d_out_ptr[idx] = d_ptr[idx] + F * (d_ptr[best_model * size + idx % size] - d_ptr[idx]) + F * (d_ptr[agent_ids[0] * size + idx % size] - d_ptr[agent_ids[1] * size + idx % size]);
			//printf("crossover layer %d of id %d with agent0 %d and agent1 %d using ri %f and R %d \n id d_ptr[%d] = %f best_model d_ptr[%d] = %f agent0 d_ptr[%d] = %f agent1 d_ptr[%d] = %f\n", layer_idx, id, agent_ids[0], agent_ids[1], ri, R,
			//	idx, d_ptr[idx], best_model * size + idx % size, d_ptr[best_model * size + idx % size], agent_ids[0] * size + idx % size, d_ptr[agent_ids[0] * size + idx % size], agent_ids[1] * size + idx % size, d_ptr[agent_ids[1] * size + idx % size]);
		}
	}
}

hiprandGenerator_t gen;
void hiprand_init() {
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 5691ULL);
	std::cout << "initializing hiprand" << std::endl;
}

int N = 64;
std::vector<std::vector<torch::Tensor>> de_crossover_cuda(const std::vector<torch::Tensor>& layers, const std::vector<torch::Tensor>& biases, int64_t NP, double CR, double F, int64_t best_model) {
	int num_layers = layers.size();
	std::vector<float*> layer_ptrs(num_layers), bias_ptrs(num_layers);
	std::vector<torch::Tensor> out_layers(num_layers), out_biases(num_layers);
	std::vector<float*> out_layer_ptrs(num_layers), out_bias_ptrs(num_layers);

	float* d_all_agent_ids;
	float* d_Rs;
	float* d_ris;
	int num_agents = NP * 3, num_Rs = NP, num_ris = num_layers * NP;
	hipMalloc(&d_all_agent_ids, num_agents * sizeof(float));
	hipMalloc(&d_Rs, num_Rs * sizeof(float));
	hipMalloc(&d_ris, num_ris * sizeof(float));

	hiprandGenerateUniform(gen, d_all_agent_ids, num_agents);
	hiprandGenerateUniform(gen, d_Rs, num_Rs);
	hiprandGenerateUniform(gen, d_ris, num_ris);
	//std::cout << "num_layers " << num_layers << std::endl;

	for (int i = 0; i < num_layers; i++) {
		// I shouldnt be copying tensors here...
		torch::Tensor layer_contig = layers[i].contiguous();
		torch::Tensor bias_contig = biases[i].contiguous();
		layer_ptrs[i] = layer_contig.data_ptr<float>();
		bias_ptrs[i] = bias_contig.data_ptr<float>();

		//out_layers[i] = torch::empty(layer_contig.sizes(), layer_contig.options());
		//out_biases[i] = torch::empty(bias_contig.sizes(), bias_contig.options());
		out_layers[i] = torch::clone(layer_contig);
		out_biases[i] = torch::clone(bias_contig);
		out_layer_ptrs[i] = out_layers[i].data_ptr<float>();
		out_bias_ptrs[i] = out_biases[i].data_ptr<float>();

		de_crossover_kernel<<<max(1l, (layer_contig.numel() + N - 1) / N), N>>>(NP, CR, F, best_model, layer_ptrs[i], out_layer_ptrs[i], layer_contig.numel() / NP, d_all_agent_ids, d_Rs, d_ris, i, num_layers);
		de_crossover_kernel<<<max(1l, (bias_contig.numel() + N - 1) / N), N>>>(NP, CR, F, best_model, bias_ptrs[i], out_bias_ptrs[i], bias_contig.numel() / NP, d_all_agent_ids, d_Rs, d_ris, i, num_layers);

		//std::cout << "layer " << i << " has " << layer_contig.numel() / NP << " parameters" << std::endl;
		//std::cout << "bias  " << i << " has " << bias_contig.numel() / NP  << " parameters" << std::endl;
	}
   	hipDeviceSynchronize();
	hipFree(d_all_agent_ids);
	hipFree(d_Rs);
	hipFree(d_ris);
 	//std::cout << "crossover finished" << std::endl;
 	return {out_layers, out_biases};
}
